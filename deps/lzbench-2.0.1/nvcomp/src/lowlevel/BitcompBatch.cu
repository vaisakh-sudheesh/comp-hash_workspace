#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "common.h"
#include "nvcomp.h"
#include "nvcomp/bitcomp.h"
#include "type_macros.h"

#ifdef ENABLE_BITCOMP
#include <bitcomp.h>

#define BTCHK(call)                                                            \
  {                                                                            \
    bitcompResult_t err = call;                                                \
    if (BITCOMP_SUCCESS != err) {                                              \
      if (err == BITCOMP_INVALID_PARAMETER)                                    \
        return nvcompErrorInvalidValue;                                        \
      else if (err == BITCOMP_INVALID_COMPRESSED_DATA)                         \
        return nvcompErrorCannotDecompress;                                    \
      else if (err == BITCOMP_INVALID_ALIGNMENT)                               \
        return nvcompErrorCannotDecompress;                                    \
      return nvcompErrorInternal;                                              \
    }                                                                          \
  }

nvcompStatus_t nvcompBatchedBitcompCompressGetMaxOutputChunkSize(
    size_t max_chunk_size,
    nvcompBatchedBitcompFormatOpts format_opts,
    size_t* max_compressed_size)
{
  *max_compressed_size = bitcompMaxBuflen(max_chunk_size);
  return nvcompSuccess;
}

nvcompStatus_t nvcompBatchedBitcompCompressAsync(
    const void* const* device_uncompressed_ptrs,
    const size_t* device_uncompressed_bytes,
    size_t, // max_uncompressed_chunk_bytes, not used
    size_t batch_size,
    void*,  // device_temp_ptr, not used
    size_t, // temp_bytes, not used
    void* const* device_compressed_ptrs,
    size_t* device_compressed_bytes,
    const nvcompBatchedBitcompFormatOpts format_opts,
    hipStream_t stream)
{
  // Convert the NVCOMP type to a BITCOMP type
  bitcompDataType_t dataType;
  switch (format_opts.data_type) {
  case NVCOMP_TYPE_CHAR:
    dataType = BITCOMP_SIGNED_8BIT;
    break;
  case NVCOMP_TYPE_USHORT:
    dataType = BITCOMP_UNSIGNED_16BIT;
    break;
  case NVCOMP_TYPE_SHORT:
    dataType = BITCOMP_SIGNED_16BIT;
    break;
  case NVCOMP_TYPE_UINT:
    dataType = BITCOMP_UNSIGNED_32BIT;
    break;
  case NVCOMP_TYPE_INT:
    dataType = BITCOMP_SIGNED_32BIT;
    break;
  case NVCOMP_TYPE_ULONGLONG:
    dataType = BITCOMP_UNSIGNED_64BIT;
    break;
  case NVCOMP_TYPE_LONGLONG:
    dataType = BITCOMP_SIGNED_64BIT;
    break;
  default:
    dataType = BITCOMP_UNSIGNED_8BIT;
  }

  // Create a Bitcomp batch handle, associate it to the stream
  bitcompAlgorithm_t algo = static_cast<bitcompAlgorithm_t>(format_opts.algorithm_type);
  bitcompHandle_t plan;
  BTCHK(bitcompCreateBatchPlan(&plan, batch_size, dataType, BITCOMP_LOSSLESS, algo));
  BTCHK(bitcompSetStream(plan, stream));

  // Launch the Bitcomp async batch compression
  BTCHK(bitcompBatchCompressLossless(
      plan,
      device_uncompressed_ptrs,
      device_compressed_ptrs,
      device_uncompressed_bytes,
      device_compressed_bytes));

  // Once launched, the handle can be destroyed
  BTCHK(bitcompDestroyPlan (plan));
  
  return nvcompSuccess;
}

// The Bitcomp batch decompression outputs bitcompResult_t statuses.
// Need to convert them to nvcompStatus_t.
__global__ void convertOutputStatuses (nvcompStatus_t *statuses, size_t batch_size)
{
  static_assert(
      sizeof(nvcompStatus_t) == sizeof(bitcompResult_t),
      "bitcomp and nvcomp statuses must be the same size");
  size_t index = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;
  if (index >= batch_size)
      return;
  bitcompResult_t ier = reinterpret_cast<bitcompResult_t *>(statuses)[index];
  nvcompStatus_t nvcomp_err = nvcompSuccess;
  if (ier != BITCOMP_SUCCESS)
  {
      if (ier == BITCOMP_INVALID_PARAMETER)
          nvcomp_err = nvcompErrorInvalidValue;
      else
          nvcomp_err = nvcompErrorCannotDecompress;
  }
  statuses[index] = nvcomp_err;
}

nvcompStatus_t nvcompBatchedBitcompDecompressAsync(
    const void* const* device_compressed_ptrs,
    const size_t*, // device_compressed_bytes, not used
    const size_t* device_uncompressed_bytes,
    size_t* device_actual_uncompressed_bytes,
    size_t batch_size,
    void* const, // device_temp_ptr, not used
    size_t,      // temp_bytes, not used
    void* const* device_uncompressed_ptrs,
    nvcompStatus_t* device_statuses,
    hipStream_t stream)
{
  // Synchronize the stream to make sure the compressed data is visible
  if (hipStreamSynchronize(stream) != hipSuccess)
    return nvcompErrorCudaError;

  // Create a Bitcomp batch handle from the compressed data.
  bitcompHandle_t plan;
  BTCHK(bitcompCreateBatchPlanFromCompressedData(&plan, device_compressed_ptrs, batch_size));

  // Associate the handle to the stream
  BTCHK(bitcompSetStream(plan, stream));

  // Launch the Bitcomp async batch decompression with extra checks
  BTCHK(bitcompBatchUncompressCheck(
      plan,
      device_compressed_ptrs,
      device_uncompressed_ptrs,
      device_uncompressed_bytes,
      (bitcompResult_t*)device_statuses));

  // Need a separate kernel to query the actual uncompressed size,
  // as bitcomp doesn't write the uncompressed size during decompression
  BTCHK(bitcompBatchGetUncompressedSizesAsync(
      device_compressed_ptrs,
      device_actual_uncompressed_bytes,
      batch_size,
      stream));

  // Also launch a kernel to convert the output statuses
  const int threads = 512;
  int blocks = (batch_size - 1) / threads + 1;
  convertOutputStatuses<<<blocks, threads, 0, stream>>>(
      device_statuses, batch_size);

  // Once launched, the handle can be destroyed
  BTCHK(bitcompDestroyPlan(plan));
  return nvcompSuccess;
}

nvcompStatus_t nvcompBatchedBitcompGetDecompressSizeAsync(
    const void* const* device_compressed_ptrs,
    const size_t* device_compressed_bytes,
    size_t* device_uncompressed_bytes,
    size_t batch_size,
    hipStream_t stream)
{
  BTCHK(bitcompBatchGetUncompressedSizesAsync(
      device_compressed_ptrs,
      device_uncompressed_bytes,
      batch_size, stream));
  return nvcompSuccess;
}

nvcompStatus_t nvcompBatchedBitcompCompressGetTempSize(
    size_t,
    size_t,
    nvcompBatchedBitcompFormatOpts,
    size_t* temp_bytes)
{
  *temp_bytes = 0;
  return nvcompSuccess;
}

nvcompStatus_t nvcompBatchedBitcompDecompressGetTempSize(
    size_t,
    size_t,
    size_t* temp_bytes)
{
  *temp_bytes = 0;
  return nvcompSuccess;
}

#else

nvcompStatus_t nvcompBatchedBitcompCompressGetMaxOutputChunkSize(
    size_t, nvcompBatchedBitcompFormatOpts, size_t*)
{
  return nvcompErrorNotSupported;
}

nvcompStatus_t nvcompBatchedBitcompCompressAsync(
    const void* const*,
    const size_t*,
    size_t,
    size_t,
    void*,
    size_t,
    void* const*,
    size_t*,
    const nvcompBatchedBitcompFormatOpts,
    hipStream_t)
{
  return nvcompErrorNotSupported;
}

nvcompStatus_t nvcompBatchedBitcompDecompressAsync(
    const void* const*,
    const size_t*,
    const size_t*,
    size_t*,
    size_t,
    void* const,
    size_t,
    void* const*,
    nvcompStatus_t*,
    hipStream_t)
{
  return nvcompErrorNotSupported;
}

nvcompStatus_t nvcompBatchedBitcompGetDecompressSizeAsync(
    const void* const*, const size_t*, size_t*, size_t, hipStream_t)
{
  return nvcompErrorNotSupported;
}

nvcompStatus_t nvcompBatchedBitcompCompressGetTempSize(
    size_t, size_t, nvcompBatchedBitcompFormatOpts, size_t*)
{
  return nvcompErrorNotSupported;
}

nvcompStatus_t nvcompBatchedBitcompDecompressGetTempSize(size_t, size_t, size_t*)
{
  return nvcompErrorNotSupported;
}

#endif
